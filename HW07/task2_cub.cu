#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <iostream>
#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
// #include "test/test_util.h"
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char *argv[]) {
    long n = atol(argv[1]);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Set up host arrays
    float* h_in;
    h_in = new float[n];
    for (long i = 0; i < n; i++) {
        h_in[i] = 1.0;
    }

    // Set up device arrays
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));
    
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
    
    // Setup device output array
    float* d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_out, sizeof(float) * n));
    
    float init = 0.0;

    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceScan::ExclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, Sum(), init, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    hipEventRecord(start);
    CubDebugExit(DeviceScan::ExclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, Sum(), init, n));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    float* h_out;
    h_out = new float[n];
    CubDebugExit(hipMemcpy(h_out, d_out, sizeof(float) * n, hipMemcpyDeviceToHost));
    
    // Check for correctness
    // printf("\t%s\n", (gpu_sum == sum ? "Test passed." : "Test falied."));
    // printf("\tSum is: %d\n", gpu_sum);
    
    //for (long i = 0; i < n; i++) {
    //	    std::cout << h_out[i] << " ";
    //}
    //std::cout << std::endl;

    std::cout << h_out[n-1] << std::endl;
    std::cout << ms << std::endl;

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    delete[] h_in;
    delete[] h_out;
    
    return 0;
}
    
