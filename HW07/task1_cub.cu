#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <iostream>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
// #include "test/test_util.h"
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Set up host arrays
    int h_in[n];
    for (int i = 0; i < n; i++) {
        h_in[i] = 1;
    }

    int  sum = 0;
    for (int i = 0; i < n; i++)
        sum += h_in[i];

    // Set up device arrays
    int* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(int) * n));
    
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * n, hipMemcpyHostToDevice));
    
    // Setup device output array
    int* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(int) * 1));
    
    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    hipEventRecord(start);
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    int gpu_sum;
    CubDebugExit(hipMemcpy(&gpu_sum, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));
    
    // Check for correctness
    // printf("\t%s\n", (gpu_sum == sum ? "Test passed." : "Test falied."));
    // printf("\tSum is: %d\n", gpu_sum);

    std::cout << gpu_sum << std::endl;
    std::cout << ms << std::endl;

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    
    return 0;
}
    