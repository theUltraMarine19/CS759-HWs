#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/functional.h>

#include <iostream>

int main(int argc, char *argv[]) {
	int n = atoi(argv[1]);

	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	thrust::host_vector<float> h_vec(n);
	for (int i = 0; i < n; i++) {
		h_vec[i] = 1.0;
	}

	thrust::device_vector<float> d_vec(h_vec.size());
	thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

	float init = 0.0;
	
	hipEventRecord(start);
	thrust::exclusive_scan(d_vec.begin(), d_vec.end(), d_vec.begin(), init, thrust::plus<float>());
	hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    std::cout << h_vec[n-1] << std::endl;
    std::cout << ms << std::endl;

	return 0;
}