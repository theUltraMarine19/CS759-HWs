#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/functional.h>

#include <iostream>

int main(int argc, char *argv[]) {
	long n = atol(argv[1]);

	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	thrust::host_vector<float> h_vec(n);
	for (long i = 0; i < n; i++) {
		h_vec[i] = 1.0;
	}

	thrust::device_vector<float> d_vec(h_vec.size());
	thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

	float init = 0.0;
	
	hipEventRecord(start);
	thrust::exclusive_scan(d_vec.begin(), d_vec.end(), d_vec.begin(), init, thrust::plus<float>());
	hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
    
    //for (long i = 0; i < n; i++) {
    //    std::cout << h_vec[i] << " ";
    //}
    //std::cout << std::endl;

    std::cout << h_vec[n-1] << std::endl;
    std::cout << ms << std::endl;

	return 0;
}
