#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <iostream>

int main(int argc, char *argv[]) {
	long n = atol(argv[1]);

	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	thrust::host_vector<int> h_vec(n);
	for (long i = 0; i < n; i++) {
		h_vec[i] = 1;
	}

	thrust::device_vector<int> d_vec(h_vec.size());
	thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
	
	int init = 0;

	hipEventRecord(start);
	int res = thrust::reduce(d_vec.begin(), d_vec.end(), init, thrust::plus<int>());
	hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << res << std::endl;
    std::cout << ms << std::endl;

	return 0;
}