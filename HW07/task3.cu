#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "count.cuh"

#include <iostream>

int main(int argc, char *argv[]) {
	long n = atol(argv[1]);

	hipEvent_t start;
  	hipEvent_t stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	thrust::host_vector<int> h_vec(n);
	// Empty output vectors
	thrust::host_vector<int> h_vals;
	thrust::host_vector<int> h_cnts;

	for (long i = 0; i < n; i++) {
		h_vec[i] = 1;
	}

	// int arr[] = {3,5,1,2,3,1};
	// h_vec = std::vector<int>(arr, arr+n);

	// for (int i = 0; i < n; i++)
	//  	std::cout << h_vec[i] << " ";
	// std::cout << std::endl;

	thrust::device_vector<int> d_vec(h_vec.size());
	thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
	// Empty device vectors
	thrust::device_vector<int> d_vals;
	thrust::device_vector<int> d_cnts;

	hipEventRecord(start);
	count(d_vec, d_vals, d_cnts);
	hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Empty host vectors will resize automatically
    h_vals = d_vals;
    // thrust::copy(d_vals.begin(), d_vals.end(), h_vals.begin());
    h_cnts = d_cnts;
    // thrust::copy(d_cnts.begin(), d_cnts.end(), h_cnts.begin());

	// for (size_t i = 0; i < h_vals.size(); i++)
	//	std::cout << h_vals[i] << " ";
	// std::cout << std::endl;

	// for (size_t i = 0; i < h_cnts.size(); i++)
	// 	std::cout << h_cnts[i] << " ";
	// std::cout << std::endl;    

    std::cout << h_vals.back() << std::endl;
    std::cout << h_cnts.back() << std::endl;
    std::cout << ms << std::endl;

	return 0;
}
