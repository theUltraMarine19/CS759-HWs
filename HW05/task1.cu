#include <iostream>
#include "reduce.cuh"
using namespace std;

int main(int argc, char* argv[]) {
  int N = atoi(argv[1]);
  int tpb = atoi(argv[2]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int* arr;
  arr = new int[N];
  for (int i = 0; i < N; i++) arr[i] = 1;

  hipEventRecord(start);
  int res = reduce(arr, N, tpb);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  // for (int i = 0; i < N; i++)
  //  	cout << out[i] << " ";
  // cout << endl;
  cout << res << endl;
  cout << ms << endl;

  free(arr);

  return 0;
}
