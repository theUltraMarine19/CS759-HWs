
#include <hip/hip_runtime.h>
__global__ void reduce_kernel(const int* g_idata, int* g_odata,
                              unsigned int n) {
  extern __shared__ int arr[];
  long tid = threadIdx.x;
  long idx = (long)blockIdx.x * (long)blockDim.x + tid;
  if (idx < n)
    arr[tid] = g_idata[idx];
  else
    arr[tid] = 0;

  __syncthreads();

  for (long i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i) {
      if (tid + i < n)  // Not needed
        arr[tid] += arr[tid + i];
    }

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = arr[0];
}

__host__ int reduce(const int* arr, unsigned int N,
                    unsigned int threads_per_block) {
  int num_blocks = (N + threads_per_block - 1) / threads_per_block;
  int *darr, *dout;
  hipMalloc((void**)&darr, N * sizeof(int));
  hipMalloc((void**)&dout, num_blocks * sizeof(int));

  // int* tmp = new int[num_blocks];
  int* tmp = new int[1];
  hipMemcpy(darr, arr, N * sizeof(int), hipMemcpyHostToDevice);

  while (num_blocks > 1) {
    reduce_kernel<<<num_blocks, threads_per_block,
                    sizeof(int) * threads_per_block>>>(darr, dout, N);
    hipDeviceSynchronize();
    // cudaMemcpy(tmp, dout, num_blocks * sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(darr, dout, num_blocks * sizeof(int), hipMemcpyDeviceToDevice);
    N = num_blocks;
    num_blocks = (num_blocks + threads_per_block - 1) / threads_per_block;
    // cudaMemcpy(darr, tmp, N * sizeof(int), cudaMemcpyHostToDevice);
  }

  reduce_kernel<<<num_blocks, threads_per_block,
                  sizeof(int) * threads_per_block>>>(darr, dout, N);
  hipDeviceSynchronize();
  hipMemcpy(tmp, dout, num_blocks * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(darr);
  hipFree(dout);

  int ret = tmp[0];
  delete[] tmp;

  // return tmp[0];
  return ret;
}
