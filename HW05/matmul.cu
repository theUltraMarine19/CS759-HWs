
#include <hip/hip_runtime.h>
#include <cstdio>
using namespace std;

__global__ void matmul_kernel(const float* A, const float* B, float* C,
                              unsigned int n) {
  extern __shared__ float arr[];
  float* sA = &arr[0];
  float* sB = &arr[blockDim.x * blockDim.y];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int aBegin = n * blockDim.x * by;
  int aEnd = aBegin + n - 1;

  int aStep = blockDim.x;

  int bBegin = blockDim.x * bx;

  int bStep = blockDim.x * n;

  float Cval = 0;
  for (int a = aBegin, b = bBegin, brow = 0; a <= aEnd;
       a += aStep, b += bStep, brow += blockDim.y) {
    int rowA = blockDim.y * by + ty;
    int colA = a + tx - blockDim.y * by * n;

    if (rowA < n && colA < n)
      sA[ty * blockDim.x + tx] = A[a + n * ty + tx];
    else
      sA[ty * blockDim.x + tx] = 0;

    int colB = blockDim.x * bx + tx;
    int rowB = brow + ty;

    if (rowB < n && colB < n)
      sB[ty * blockDim.x + tx] = B[b + n * ty + tx];
    else
      sB[ty * blockDim.x + tx] = 0;

    __syncthreads();

    // if (tx == 0 && ty == 0 && bx == 0 && by == 0) {
    // 	for (int i = 0; i < blockDim.y; i++) {
    // 		for (int j = 0; j < blockDim.x; j++) {
    // 			printf("%f ", sA[i*blockDim.x + j]);
    // 		}
    // 		printf("\n");
    // 	}
    // 	printf("---------\n");

    // 	for (int i = 0; i < blockDim.y; i++) {
    // 		for (int j = 0; j < blockDim.x; j++) {
    // 			printf("%f ", sB[i*blockDim.x + j]);
    // 		}
    // 		printf("\n");
    // 	}
    // }

    // safe because out-of-bounds entries are 0
    for (int k = 0; k < blockDim.x; k++) {
      Cval += sA[ty * blockDim.x + k] * sB[k * blockDim.x + tx];
    }

    // if (tx == 1 && ty == 1)
    // 	printf("%f\n", Cval);

    __syncthreads();
  }

  int idx = n * blockDim.x * by + blockDim.x * bx;
  int rowC = blockDim.y * by + ty;
  int colC = blockDim.x * bx + tx;
  if (rowC < n && colC < n) C[idx + n * ty + tx] = Cval;
}

__host__ void matmul(const float* A, const float* B, float* C, unsigned int n,
                     unsigned int block_dim) {
  dim3 block(block_dim, block_dim);
  dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);
  matmul_kernel<<<grid, block, 2 * sizeof(float) * block_dim * block_dim>>>(
      A, B, C, n);
  hipDeviceSynchronize();
}