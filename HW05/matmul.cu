
#include <hip/hip_runtime.h>
__global__ void matmul_kernel(const float* A, const float* B, float* C, unsigned int n) {
	extern __shared__ float arr[];
	float *sA = &arr[0];
	float* sB = &arr[blockDim.x*blockDim.y];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = n*blockDim.x*by;
	int aEnd = aBegin+n-1;

	int aStep = blockDim.x;

	int bBegin = blockDim.x * bx;

	int bStep = blockDim.x * n;

	float Cval = 0;
	for (int a = aBegin, b = bBegin; a <= aEnd; a+= aStep, b+= bStep) {
		int rowA = (a+n*ty + tx)/n;
		int colA = (a+n*ty + tx)%n;
		
		if (rowA < n && colA < n)
			sA[ty*blockDim.x + tx] = A[a+n*ty + tx];
		else
			sA[ty*blockDim.x + tx] = 0;

		int rowB = (b+n*ty+tx)/n;
		int colB = (b+n*ty+tx)%n;
		
		if (rowB < n && colB < n)
			sB[ty*blockDim.x + tx] = B[b+n*ty+tx];
		else
			sB[ty*blockDim.x + tx] = 0;

		__syncthreads();

		// safe because out-of-bounds entries are 0
		for (int k = 0; k < blockDim.x; k++) {
			Cval += sA[ty*blockDim.x + k] * sB[k*blockDim.x+tx];
		}

		__syncthreads();
	}

	int idx = n*blockDim.x*by + blockDim.x*bx;
	int rowC = (idx + n*ty+tx)/n;
	int colC = (idx + n*ty+tx)%n;
	if (rowC < n && colC < n)
		C[idx + n*ty+tx] = Cval;
}

__host__ void matmul(const float* A, const float* B, float* C, unsigned int n, unsigned int block_dim) {
	dim3 block(block_dim, block_dim);
	dim3 grid((n+block.x-1)/block.x, (n+block.y-1)/block.y);
	matmul_kernel<<<grid, block, 2*sizeof(float)*block_dim*block_dim>>>(A, B, C, n);
	hipDeviceSynchronize();
}