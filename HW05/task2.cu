#include <iostream>
#include "matmul.cuh"
using namespace std;

int main(int argc, char* argv[]) {
	int n = atoi(argv[1]);
	int block_dim = atoi(argv[2]);
	
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* A, *B, *C;
	hipMallocManaged(&A, n * n * sizeof(float));
  	hipMallocManaged(&B, n * n * sizeof(float));
  	hipMallocManaged(&C, n * n * sizeof(float));

	for (long i = 0; i < n*n; i++) {
		A[i] = 1.0;
		B[i] = 1.0;
		C[i] = -1.0;
	}

  	hipEventRecord(start);
  	matmul(A, B, C, n, block_dim);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	// for (int i = 0; i < n; i++)
  	// 	cout << out[i] << " ";
  	// cout << endl;	
  	cout << C[0] << endl;
  	cout << C[n*n-1] << endl;
  	cout << ms << endl;

  	hipFree(A);
  	hipFree(B);
  	hipFree(C);
  	
  	return 0;
}
