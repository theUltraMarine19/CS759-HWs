#include <iostream>
#include "matmul.cuh"
using namespace std;

int main(int argc, char* argv[]) {
  int n = atoi(argv[1]);
  int block_dim = atoi(argv[2]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  for (long i = 0; i < n * n; i++) {
    // A[i] = i+1;
    // B[i] = n*n-i;
    A[i] = 0.5;
    B[i] = 0.5;
    C[i] = -1.0;
  }

  hipEventRecord(start);
  matmul(A, B, C, n, block_dim);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  // for (int i = 0; i < n*n; i++)
  // 	cout << C[i] << " ";
  // cout << endl;
  cout << C[0] << endl;
  cout << C[n * n - 1] << endl;
  cout << ms << endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}
