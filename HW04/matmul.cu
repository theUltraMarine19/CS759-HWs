
#include <hip/hip_runtime.h>
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n*n) {
		C[idx] = 0;
		int row = idx/n;
		int col = idx%n;
		for (int k = 0; k < n; k++)
			C[idx] += A[row*n+k] * B[k*n+col];
	}
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {
	int num_blocks = (n*n + threads_per_block - 1)/threads_per_block;

	matmul_kernel<<<num_blocks, threads_per_block>>>(A, B, C, n);
	hipDeviceSynchronize();

}