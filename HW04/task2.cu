#include <iostream>
#include "stencil.cuh"
using namespace std;

int main(int argc, char* argv[]) {
	long n = atol(argv[1]);
	int R = atoi(argv[2]);
	int tpb = atoi(argv[3]);
	
	// int dev;
	// hipDeviceProp_t prop;
	// hipGetDevice(&dev);
	// hipGetDeviceProperties(&prop, dev);
	// cout << prop.sharedMemPerBlock << endl;
	
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* img, *out, *mask;
	img = new float[n];
	out = new float[n];
	mask = new float[2*R+1];

	for (long i = 0; i < n; i++) {
		// img[i] = i;
		img[i] = 1.0;
		// out[i] = 0.0;
	}

	for (int i = 0; i < 2*R+1; i++) {
		// mask[i] = i+1;
		mask[i] = 1.0;
	}

	float *dimg, *dout, *dmask;
	hipMallocManaged((void **)&dimg, n * sizeof(float));
  	hipMallocManaged((void **)&dout, n * sizeof(float));
  	hipMallocManaged((void **)&dmask, (2*R+1) * sizeof(float));

  	hipMemcpy(dimg, img, n * sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(dmask, mask, (2*R+1) * sizeof(float), hipMemcpyHostToDevice);

  	hipEventRecord(start);
  	stencil(dimg, dmask, dout, n, R, tpb);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	hipMemcpy(out, dout, n * sizeof(float), hipMemcpyDeviceToHost);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);

  	// for (int i = 0; i < n; i++)
  	// 	cout << out[i] << " ";
  	// cout << endl;	
  	cout << out[n-1] << endl;
  	cout << ms << endl;

  	hipFree(dimg);
  	hipFree(dout);
  	hipFree(dmask);
  	free(img);
  	free(out);
  	free(mask);
  	
  	return 0;
}
