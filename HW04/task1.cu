#include <iostream>
#include "matmul.cuh"
using namespace std;

int main(int argc, char* argv[]) {
	int n = atoi(argv[1]);
	int tpb = atoi(argv[2]);

    // cout << n << endl;

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* a = new float[n*n];
	float* b = new float[n*n];

	for (int i = 0; i < n*n; i++) {
		// a[i] = i+1;
		// b[i] = (n*n-i);
        a[i] = 1.0;
        b[i] = 1.0;
	}

    // a = {3, 1, 2, 6, 8, 7, 9, 4, 3};
    // b = {1, 2, 3, 4, 5, 6, 7, 8, 9};

	float* c = new float[n*n];

	float* dA, *dB, *dC;
	hipMallocManaged((void **)&dA, n*n * sizeof(float));
  	hipMallocManaged((void **)&dB, n*n * sizeof(float));
  	hipMallocManaged((void **)&dC, n*n * sizeof(float));

  	hipMemcpy(dA, a, n*n * sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(dB, b, n*n * sizeof(float), hipMemcpyHostToDevice);

  	hipEventRecord(start);
  	matmul(dA, dB, dC, n, tpb);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	hipMemcpy(c, dC, n*n * sizeof(float), hipMemcpyDeviceToHost);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);
  	
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++)
    //         cout << c[i*n+j] << " ";
    //     cout << endl;
    // }

  	cout << c[n*n-1] << endl;
  	cout << ms << endl;

  	hipFree(dA);
  	hipFree(dB);
  	hipFree(dC);
  	free(a);
  	free(b);
  	free(c);
  	
  	return 0;

}