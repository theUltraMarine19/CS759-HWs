#include <iostream>
#include "matmul.cuh"
using namespace std;

int main(int argc, char* argv[]) {
	int n = atoi(argv[1]);
	int tpb = atoi(argv[2]);

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* a = new float[n*n];
	float* b = new float[n*n];

	for (int i = 0; i < n*n; i++) {
		a[i] = 1.0;
		b[i] = 1.0;
	}

	float* c = new float[n*n];

	float* dA, *dB, *dC;
	hipMallocManaged((void **)&dA, n*n * sizeof(float));
  	hipMallocManaged((void **)&dB, n*n * sizeof(float));
  	hipMallocManaged((void **)&dC, n*n * sizeof(float));

  	hipMemcpy(dA, a, n*n * sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(dB, b, n*n * sizeof(float), hipMemcpyHostToDevice);

  	hipEventRecord(start);
  	matmul(dA, dB, dC, n, tpb);
  	hipEventRecord(stop);
  	hipEventSynchronize(stop);

  	hipMemcpy(c, dC, n*n * sizeof(float), hipMemcpyDeviceToHost);

  	float ms;
  	hipEventElapsedTime(&ms, start, stop);
  	
  	cout << c[n*n-1] << endl;
  	cout << ms << endl;

  	hipFree(dA);
  	hipFree(dB);
  	hipFree(dC);
  	free(a);
  	free(b);
  	free(c);
  	
  	return 0;

}