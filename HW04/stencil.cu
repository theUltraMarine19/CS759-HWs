#include "hip/hip_runtime.h"
#include <cstdio>
#include "stencil.cuh"
using namespace std;

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int block_size = blockDim.x;

	extern __shared__ float arr[];
	float* img = &arr[0]; 									// block_size + 2*R 
	float* msk = &arr[2*R + block_size]; 
	float* out = &arr[block_size + 4*R + 1]; 				// block_size

	long idx = tidx + (long)block_size * (long)bidx; 		// long since can be > 2^31 -1
	int curr = tidx + R;

	// if (tidx == block_size-1 || tidx == 0)
	//  	printf("Hello %ld %d %d\n", idx+R, curr-R, curr+R);

	if (idx < n)
		img[curr] = image[idx];
	else
		img[curr] = 0;

	if (tidx < 2*R+1)
		msk[tidx] = mask[tidx];

	if (tidx < R) {
		
		if (idx >= R)
			img[curr-R] = image[idx-R];
		else
			img[curr-R] = 0;
		
	}

	if (tidx + R >= block_size) {

		if (idx+R < n) 
			img[curr+R] = image[idx + R];	
		else
			img[curr+R] = 0;
		
	}

	__syncthreads();

	// if (tidx == 0 && bidx == 0) {
	// 	printf("------\n"); 
	// 	for (int i = 0; i < block_size; i++)
	// 		printf("%f ", img[i]);
	// 	printf("\n----------\n");
	// }

	out[tidx] = 0;
	for (int i = 0; i <= 2*R; i++) {
		// printf("%d ", i);
		// if (tidx == 1 && bidx == 0) {
		// 	printf("%f %f %f \n", img[curr+i-R], msk[i], out[tidx]);
		// }
		// long img_idx = (long)curr+i;
		out[tidx] += img[curr+i-R] * msk[i];
	}

	__syncthreads();

	if (idx < n)
		output[idx] = out[tidx];

}

__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block) {

	int num_blocks = ((long)n+(long)threads_per_block-1)/((long)threads_per_block);
	stencil_kernel<<<num_blocks, threads_per_block, sizeof(float)*(threads_per_block + 2*R) + sizeof(float)*(2*R+1) + sizeof(float)*(threads_per_block)>>>(image, mask, output, n, R);
	hipDeviceSynchronize();

}
