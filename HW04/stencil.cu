#include "hip/hip_runtime.h"
#include <cstdio>
#include "stencil.cuh"
using namespace std;

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int block_size = blockDim.x;

	

	extern __shared__ float arr[];
	float* img = &arr[0]; // block_size + 2*R 
	float* msk = &arr[2*R + block_size]; 
	float* out = &arr[block_size + 4*R + 1]; // block_size

	// if (tidx == 1 && bidx == 0)
	// printf("Hello %d %d\n", bidx, tidx);

	long idx = tidx + block_size * bidx;
	int curr = tidx + R;


	img[curr] = image[idx];

	if (tidx < 2*R+1)
		msk[tidx] = mask[tidx];

	if (tidx < R) {
		if (idx >= R)
			img[curr-R] = image[idx-R];
		else
			img[curr-R] = 0;
		if (idx + block_size < n)
			img[curr + block_size] = image[idx + block_size];
		else
			img[curr + block_size] = 0;
	}

	__syncthreads();

	// if (tidx == 0 && bidx == 0) {
	// 	printf("------\n"); 
	// 	for (int i = 0; i < block_size; i++)
	// 		printf("%f ", img[i]);
	// 	printf("\n----------\n");
	// }

	out[tidx] = 0;
	for (int i = 0; i <= 2*R; i++) {
		// printf("%d ", i);
		// if (tidx == 1 && bidx == 0) {
		// 	printf("%f %f %f \n", img[curr+i-R], msk[i], out[tidx]);
		// }
		out[tidx] += img[curr+i-R] * msk[i];
	}

	__syncthreads();

	output[idx] = out[tidx];

}

__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block) {

	int num_blocks = (n+threads_per_block-1)/threads_per_block;
	stencil_kernel<<<num_blocks, threads_per_block, sizeof(float)*(threads_per_block + 2*R) + sizeof(float)*(2*R+1) + sizeof(float)*(threads_per_block)>>>(image, mask, output, n, R);
	hipDeviceSynchronize();

}