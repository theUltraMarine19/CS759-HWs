#include <cstdio>
#include "vadd.cuh"
#define THREADS_PER_BLOCK 512
using namespace std;

int main(int argc, char *argv[]) {
  int n = atoi(argv[1]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *ha, *hb;
  ha = new float[n];
  hb = new float[n];

  for (int i = 0; i < n; i++) {
    ha[i] = 1.0 * (i + 1);
    hb[i] = 1.0 * (i + 1);
  }

  float *da, *db;
  hipMalloc((void **)&da, n * sizeof(float));
  hipMalloc((void **)&db, n * sizeof(float));

  hipMemcpy(da, ha, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(db, hb, n * sizeof(float), hipMemcpyHostToDevice);

  int num_blocks = (n - 1) / THREADS_PER_BLOCK + 1;

  hipEventRecord(start);
  vadd<<<num_blocks, THREADS_PER_BLOCK>>>(da, db, n);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  printf("%f\n", ms / 1000.0);

  hipMemcpy(hb, db, n * sizeof(float), hipMemcpyDeviceToHost);

  printf("%f\n%f\n", hb[0], hb[n - 1]);

  free(ha);
  free(hb);
  hipFree(da);
  hipFree(db);

  return 0;
}
