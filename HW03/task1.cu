
#include <hip/hip_runtime.h>
#include<cstdio>
using namespace std;

__global__ void printThread(void) {
	printf("Hello World! I am thread %d", threadIdx.x);
}

int main() {
	printThread<<<1,4>>>();
	hipDeviceSynchronize();
	return 0;
}