
#include <hip/hip_runtime.h>
#include <cstdio>
#define N 16
#define THREADS_PER_BLOCK 8
using namespace std;

__global__ void add(int *dA) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  dA[idx] = threadIdx.x + blockIdx.x;
}

int main() {
  int *dA;
  int size = N * sizeof(int);
  hipMalloc((void **)&dA, size);

  int *hA;
  hA = new int[size];

  add<<<2, THREADS_PER_BLOCK>>>(dA);

  hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++)
    if (i == N - 1)
      printf("%d", hA[i]);
    else
      printf("%d ", hA[i]);

  free(hA);
  hipFree(dA);
  return 0;
}
