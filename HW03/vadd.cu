
#include <hip/hip_runtime.h>
__global__ void vadd(const float *a, float *b, unsigned int n) {
	int idx = threadIdx.x + blockIdx.x* blockDim.x;
	if (idx < n) {
		b[idx] += a[idx];
	}
}