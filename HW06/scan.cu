
#include <hip/hip_runtime.h>
#include <cstdio>
using namespace std;

__global__ void adder(float* arr, float* block_incrs, int n) {
  // int tid = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;

  float add_val = block_incrs[blockIdx.x];

  if (gtid < n) arr[gtid] += add_val;
}

__global__ void hillis_steele(float* g_idata, float* g_odata, int n,
                              float* block_sums) {
  // all memory writes to be serviced immediately
  extern volatile __shared__ float temp[];

  int tid = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  int block_size = blockDim.x;

  int pout = 0, pin = 1;
  float last_element;

  // load input into shared memory
  if (tid == 0 || gtid >= n)
    temp[tid] = 0;
  else
    temp[tid] = g_idata[gtid - 1];

  if (tid == block_size - 1)
    if (gtid < n)
      last_element = g_idata[gtid];
    else
      last_element = 0.0;

  // entire temp should've gotten populated
  __syncthreads();

  // if (gtid == 5) {
  // 	std::printf("global : %f===============\n", g_idata[gtid]);
  //    	std::printf("temp : %f===============\n", temp[tid]);
  // }

  for (int offset = 1; offset < block_size; offset *= 2) {
    pout = 1 - pout;  // swap double buffer indices
    pin = 1 - pout;

    if (tid >= offset)
      temp[pout * block_size + tid] =
          temp[pin * block_size + tid] + temp[pin * block_size + tid - offset];
    else
      temp[pout * block_size + tid] = temp[pin * block_size + tid];

    __syncthreads();  // I need this here before I start next iteration
  }

  if (gtid < n) g_odata[gtid] = temp[pout * block_size + tid];

  if (tid == block_size - 1)
    block_sums[blockIdx.x] = last_element + temp[pout * block_size + tid];
}

__host__ void scan(const float* in, float* out, unsigned int n,
                   unsigned int threads_per_block) {
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  // printf("num blocks : %d\n", num_blocks);

  float *din, *dout, *block_sums, *block_incrs, *dummy;
  hipMalloc((void**)&din, n * sizeof(float));
  hipMalloc((void**)&dout, n * sizeof(float));
  hipMallocManaged((void**)&block_sums, num_blocks * sizeof(float));
  hipMallocManaged((void**)&block_incrs, num_blocks * sizeof(float));
  hipMallocManaged((void**)&dummy, sizeof(float));

  hipMemcpy(din, in, n * sizeof(float), hipMemcpyHostToHost);

  // // Only applicable if threads_per_block is a power of 2
  // reduce_kernel<<<num_blocks, threads_per_block, sizeof(float) *
  // threads_per_block>>>(din, block_sums, n);

  hillis_steele<<<num_blocks, threads_per_block,
                  2 * threads_per_block * sizeof(float)>>>(din, dout, n,
                                                           block_sums);
  hipDeviceSynchronize();

  // get the block increments (scan it once because of assumption)
  int new_num_blocks = (num_blocks + threads_per_block - 1) /
                       threads_per_block;  // will always be 1
  hillis_steele<<<new_num_blocks, threads_per_block,
                  2 * threads_per_block * sizeof(float)>>>(
      block_sums, block_incrs, num_blocks, dummy);
  hipDeviceSynchronize();

  // for (int i = 0; i < num_blocks; i++)
  // 	printf("%f ", block_incrs[i]);
  // printf("\n");

  // add each block increment to each block
  adder<<<num_blocks, threads_per_block>>>(dout, block_incrs, n);
  hipDeviceSynchronize();

  hipMemcpy(out, dout, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(din);
  hipFree(dout);
  hipFree(block_sums);
  hipFree(block_incrs);
  hipFree(dummy);
}