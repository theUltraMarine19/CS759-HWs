#include <hipblas.h>
#include <iostream>
#include "mmul.h"
using namespace std;

int main(int argc, char* argv[]) {
  int n = atoi(argv[1]);
  int n_tests = atoi(argv[2]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  float tot = 0;

  // these are column-major
  for (long i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      //A[j*n+i] = 1.0;
      //B[j*n+i] = 1.0;
      A[j*n+i] = i*n+j;
      B[j*n+i] = (n-i-1)*n + (n-j-1);
      C[j*n+i] = -1.0;
    }
  }

  for (int i = 0; i < n_tests; i++) {
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    for (long i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        C[j*n+i] = -1.0;
      }
    }
    hipEventRecord(start);
    mmul(handle, A, B, C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    tot += ms;
    // for (int i = 0; i < n; i++)
    //  for (int j = 0; j < n; j++)
    //     cout << C[j*n+i] << " ";
    // cout << endl;  
  }

  cout << tot / n_tests << endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);

  return 0;
}
