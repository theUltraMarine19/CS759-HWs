#include <hipblas.h>
#include <iostream>
#include "mmul.h"
using namespace std;

int main(int argc, char* argv[]) {
  int n = atoi(argv[1]);
  int n_tests = atoi(argv[2]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  // these are column-major
  for (long i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
    // B[i] = (i+1)*0.1;
    // A[i] = (n*n-i-1)*0.1;
      A[j*n+i] = i*n+j;
      B[j*n+i] = (n-i-1)*n + (n-j-1);
      C[j*n+i] = -1.0;
    }
  }

  hipEventRecord(start);

  for (int i = 0; i < n_tests; i++) {
    // hipblasSetMathMode(handle, CUBLAS TENSOR OP MATH);
    mmul(handle, A, B, C, n);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
   	  cout << C[j*n+i] << " ";
  cout << endl;
  cout << ms / n_tests << endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);

  return 0;
}
