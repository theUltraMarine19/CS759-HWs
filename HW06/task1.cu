#include <hipblas.h>
#include <iostream>
#include "mmul.h"
using namespace std;

int main(int argc, char* argv[]) {
  int n = atoi(argv[1]);
  int n_tests = atoi(argv[2]);

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float *A, *B, *C;
  hipMallocManaged(&A, n * n * sizeof(float));
  hipMallocManaged(&B, n * n * sizeof(float));
  hipMallocManaged(&C, n * n * sizeof(float));

  // these are column-major
  for (long i = 0; i < n * n; i++) {
    // B[i] = (i+1)*0.1;
    // A[i] = (n*n-i-1)*0.1;
    A[i] = 0.5;
    B[i] = 0.5;
    C[i] = -1.0;
  }

  hipEventRecord(start);

  for (int i = 0; i < n_tests; i++) {
    // hipblasSetMathMode(handle, CUBLAS TENSOR OP MATH);
    mmul(handle, A, B, C, n);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  // for (int i = 0; i < n*n; i++)
  //  	cout << C[i] << " ";
  // cout << endl;
  cout << ms / n_tests << endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);

  return 0;
}
