#include <hipblas.h>
#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C,
          int n) {
  float alpha = 1, beta = 1;
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n,
              &beta, C, n);
  hipDeviceSynchronize();
}