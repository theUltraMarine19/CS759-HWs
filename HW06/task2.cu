#include <iostream>
#include "scan.cuh"
using namespace std;

int main(int argc, char* argv[]) {
  long N = atol(argv[1]);
  // int tpb = atoi(argv[2]);
  int tpb = 1024;

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* arr;
  arr = new float[N];
  float* out;
  out = new float[N];
  for (long i = 0; i < N; i++) {
	  arr[i] = 1.0;
	  out[i] = 0.0;
  }

  hipEventRecord(start);
  scan(arr, out, N, tpb);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  //for (int i = 0; i < N; i++)
  //  	cout << out[i] << " ";
  //cout << endl;

  cout << out[N - 1] << endl;
  cout << ms << endl;

  delete[] arr;
  delete[] out;

  return 0;
}
