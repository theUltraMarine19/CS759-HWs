#include <iostream>
#include "scan.cuh"
using namespace std;

int main(int argc, char* argv[]) {
  int N = atoi(argv[1]);
  // int tpb = atoi(argv[2]);
  int tpb = 1024;

  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* arr;
  arr = new float[N];
  float* out;
  out = new float[N];
  for (int i = 0; i < N; i++) {
	  arr[i] = 1;
	  out[i] = 0;
  }

  hipEventRecord(start);
  scan(arr, out, N, tpb);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float ms;
  hipEventElapsedTime(&ms, start, stop);

  //for (int i = 0; i < N; i++)
  //  	cout << out[i] << " ";
  //cout << endl;

  cout << out[N - 1] << endl;
  cout << ms << endl;

  delete[] arr;
  delete[] out;

  return 0;
}
